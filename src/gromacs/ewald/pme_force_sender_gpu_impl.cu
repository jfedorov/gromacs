/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements PME-PP communication using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_force_sender_gpu_impl.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

/*! \brief Create PME-PP GPU communication object */
PmeForceSenderGpu::Impl::Impl(GpuEventSynchronizer*  pmeForcesReady,
                              MPI_Comm               comm,
                              gmx::ArrayRef<PpRanks> ppRanks) :
    pmeForcesReady_(pmeForcesReady), comm_(comm), ppRanks_(ppRanks)
{
}

PmeForceSenderGpu::Impl::~Impl() = default;

/*! \brief  sends force buffer address to PP ranks */
void PmeForceSenderGpu::Impl::sendForceBufferAddressToPpRanks(DeviceBuffer<Float3> d_f)
{
    // Need to send address to PP rank only for thread-MPI as PP rank pulls
    // data using cudamemcpy
    if (!GMX_THREAD_MPI)
    {
        return;
    }
#if GMX_MPI
    int ind_start = 0;
    int ind_end   = 0;
    for (const auto& receiver : ppRanks_)
    {
        ind_start = ind_end;
        ind_end   = ind_start + receiver.numAtoms;

        // Data will be transferred directly from GPU.
        Float3* sendBuf = &d_f[ind_start];

        MPI_Send(&sendBuf, sizeof(Float3*), MPI_BYTE, receiver.rankId, 0, comm_);
    }
#else
    GMX_UNUSED_VALUE(d_f);
#endif
}

/*! \brief Send PME synchronizer directly using CUDA memory copy */
void PmeForceSenderGpu::Impl::sendFSynchronizerToPpCudaDirect(int ppRank)
{
    GMX_ASSERT(GMX_THREAD_MPI,
               "sendFSynchronizerToPpCudaDirect is expected to be called only for Thread-MPI");

    // Data will be pulled directly from PP task
#if GMX_MPI
    // TODO Using MPI_Isend would be more efficient, particularly when
    // sending to multiple PP ranks
    // NOLINTNEXTLINE(bugprone-sizeof-expression)
    MPI_Send(&pmeForcesReady_, sizeof(GpuEventSynchronizer*), MPI_BYTE, ppRank, 0, comm_);
#else
    GMX_UNUSED_VALUE(ppRank);
#endif
}

/*! \brief Send PME data directly using CUDA-aware MPI */
void PmeForceSenderGpu::Impl::sendFToPpCudaMpi(DeviceBuffer<RVec> sendbuf,
                                               int                offset,
                                               int                numBytes,
                                               int                ppRank,
                                               MPI_Request*       request)
{
    GMX_ASSERT(GMX_LIB_MPI, "sendFToPpCudaMpi is expected to be called only for Lib-MPI");

#if GMX_MPI
    // if using GPU direct comm with CUDA-aware MPI, make sure forces are ready on device
    // before sending it to PP ranks
    pmeForcesReady_->waitForEvent();

    MPI_Isend(sendbuf[offset], numBytes, MPI_BYTE, ppRank, 0, comm_, request);

#else
    GMX_UNUSED_VALUE(sendbuf);
    GMX_UNUSED_VALUE(offset);
    GMX_UNUSED_VALUE(numBytes);
    GMX_UNUSED_VALUE(ppRank);
    GMX_UNUSED_VALUE(request);
#endif
}

PmeForceSenderGpu::PmeForceSenderGpu(GpuEventSynchronizer*  pmeForcesReady,
                                     MPI_Comm               comm,
                                     gmx::ArrayRef<PpRanks> ppRanks) :
    impl_(new Impl(pmeForcesReady, comm, ppRanks))
{
}

PmeForceSenderGpu::~PmeForceSenderGpu() = default;

void PmeForceSenderGpu::sendForceBufferAddressToPpRanks(DeviceBuffer<RVec> d_f)
{
    impl_->sendForceBufferAddressToPpRanks(d_f);
}

void PmeForceSenderGpu::sendFSynchronizerToPpCudaDirect(int ppRank)
{
    impl_->sendFSynchronizerToPpCudaDirect(ppRank);
}

void PmeForceSenderGpu::sendFToPpCudaMpi(DeviceBuffer<RVec> sendbuf,
                                         int                offset,
                                         int                numBytes,
                                         int                ppRank,
                                         MPI_Request*       request)
{
    impl_->sendFToPpCudaMpi(sendbuf, offset, numBytes, ppRank, request);
}


} // namespace gmx
