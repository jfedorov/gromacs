/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019,2020, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements PME-PP communication using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_pp_comm_gpu_impl.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/device_context.h"
#include "gromacs/gpu_utils/device_stream.h"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

PmePpCommGpu::Impl::Impl(MPI_Comm             comm,
                         int                  pmeRank,
                         const DeviceContext& deviceContext,
                         const DeviceStream&  deviceStream) :
    deviceContext_(deviceContext),
    comm_(comm),
    pmeRank_(pmeRank),
    status_(MPI_REQUEST_NULL)
#if GMX_THREAD_MPI
    ,
    pmePpCommStream_(deviceStream)
#endif
{
}

PmePpCommGpu::Impl::~Impl()
{
#if GMX_LIB_MPI
    // resource clean-up as MPI_waitcall might not be called on this status_
    if (status_ != MPI_REQUEST_NULL)
        MPI_Request_free(&status_);

    // free staging buffer on GPU. This code is workaround for UCX bug
    // https://github.com/openucx/ucx/issues/4722
    freeDeviceBuffer(d_ppCoord_);
#endif
}

void PmePpCommGpu::Impl::reinit(int size)
{
    // This rank will access PME rank memory directly, so needs to receive the remote PME buffer addresses.
#if GMX_MPI

#    if GMX_THREAD_MPI

    MPI_Recv(&remotePmeXBuffer_, sizeof(void**), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
    MPI_Recv(&remotePmeFBuffer_, sizeof(void**), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
#    else
    // Reallocate buffer used for staging PP co-ordinates on GPU. This is needed only for process-MPI
    // as UCX layer has bug due to which host->device data trasnfer seg faults inside UCX layer.
    // Bug: https://github.com/openucx/ucx/issues/4722
    // ToDo: Evaluate if we really need to create new staging area or some already existing memory can be used
    // like stateGpu->getCoordinates()
    reallocateDeviceBuffer(&d_ppCoord_, size, &d_ppCoordSize_, &d_ppCoordSizeAlloc_, deviceContext_);
#    endif

    // Reallocate buffer used for staging PME force on GPU
    reallocateDeviceBuffer(&d_pmeForces_, size, &d_pmeForcesSize_, &d_pmeForcesSizeAlloc_, deviceContext_);
#else
    GMX_UNUSED_VALUE(size);
#endif
    return;
}

void PmePpCommGpu::Impl::receiveForceFromPme(void* recvPtr, int recvSize, bool receivePmeForceToGpu)
{
#if GMX_MPI
    void* pmeForcePtr = receivePmeForceToGpu ? static_cast<void*>(d_pmeForces_) : recvPtr;

#    if GMX_THREAD_MPI
    // Receive event from PME task and add to stream, to ensure pull of data doesn't
    // occur before PME force calc is completed
    GpuEventSynchronizer* pmeSync;
    MPI_Recv(&pmeSync, sizeof(GpuEventSynchronizer*), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
    pmeSync->enqueueWaitEvent(pmePpCommStream_);

    // Pull force data from remote GPU
    hipError_t stat = hipMemcpyAsync(pmeForcePtr, remotePmeFBuffer_, recvSize * DIM * sizeof(float),
                                       hipMemcpyDefault, pmePpCommStream_.stream());
    CU_RET_ERR(stat, "hipMemcpyAsync on Recv from PME CUDA direct data transfer failed");

    if (receivePmeForceToGpu)
    {
        // Record event to be enqueued in the GPU local buffer operations, to
        // satisfy dependency on receiving the PME force data before
        // reducing it with the other force contributions.
        forcesReadySynchronizer_.markEvent(pmePpCommStream_);
    }
    else
    {
        // Ensure CPU waits for PME forces to be copied before reducing
        // them with other forces on the CPU
        hipStreamSynchronize(pmePpCommStream_.stream());
    }
#    else
    if (status_ != MPI_REQUEST_NULL)
    {
        // Resource clean-up
        MPI_Request_free(&status_);
        status_ = MPI_REQUEST_NULL;
    }

    MPI_Irecv(pmeForcePtr, recvSize * DIM, MPI_FLOAT, pmeRank_, 0, comm_, &status_);

    if (!receivePmeForceToGpu)
    {
        // Need to wait as host pointer is expected to be used right after this call
        MPI_Wait(&status_, MPI_STATUS_IGNORE);
    }

#    endif // GMX_THREAD_MPI

#else
    GMX_UNUSED_VALUE(recvPtr);
    GMX_UNUSED_VALUE(recvSize);
    GMX_UNUSED_VALUE(receivePmeForceToGpu);
#endif
}

#if GMX_MPI
#    if GMX_THREAD_MPI
void PmePpCommGpu::Impl::sendCoordinatesToPmeCudaDirect(void* sendPtr,
                                                        int   sendSize,
                                                        bool gmx_unused sendPmeCoordinatesFromGpu,
                                                        GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    // ensure stream waits until coordinate data is available on device
    coordinatesReadyOnDeviceEvent->enqueueWaitEvent(pmePpCommStream_);

    hipError_t stat = hipMemcpyAsync(remotePmeXBuffer_, sendPtr, sendSize * DIM * sizeof(float),
                                       hipMemcpyDefault, pmePpCommStream_.stream());
    CU_RET_ERR(stat, "hipMemcpyAsync on Send to PME CUDA direct data transfer failed");

    // Record and send event to allow PME task to sync to above transfer before commencing force calculations
    pmeCoordinatesSynchronizer_.markEvent(pmePpCommStream_);
    GpuEventSynchronizer* pmeSync = &pmeCoordinatesSynchronizer_;
    MPI_Send(&pmeSync, sizeof(GpuEventSynchronizer*), MPI_BYTE, pmeRank_, 0, comm_);
}

#    else

void PmePpCommGpu::Impl::sendCoordinatesToPmeCudaMPI(void* sendPtr,
                                                     int sendSize,
                                                     bool gmx_unused sendPmeCoordinatesFromGpu,
                                                     GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    // ensure coordinate data is available on device before we start transfer
    coordinatesReadyOnDeviceEvent->waitForEvent();

    void* sendptr_x = sendPtr;
    if (!sendPmeCoordinatesFromGpu)
    {
        // need an explcit copy as UCX has a bug due to which sending host buffer
        // to a device buffer cause crash inside UCX. This has been reported to UCX team.
        hipError_t stat =
                hipMemcpy(d_ppCoord_, sendPtr, sendSize * DIM * sizeof(float), hipMemcpyHostToDevice);
        CU_RET_ERR(stat, "hipMemcpy on Send to PME CUDA data transfer failed");

        sendptr_x = d_ppCoord_;
    }

    MPI_Send(sendptr_x, sendSize * DIM, MPI_FLOAT, pmeRank_, 0, comm_);
}
#    endif
#endif

void PmePpCommGpu::Impl::sendCoordinatesToPme(void* sendPtr,
                                              int   sendSize,
                                              bool gmx_unused       sendPmeCoordinatesFromGpu,
                                              GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
#if GMX_MPI

#    if GMX_THREAD_MPI
    sendCoordinatesToPmeCudaDirect(sendPtr, sendSize, sendPmeCoordinatesFromGpu,
                                   coordinatesReadyOnDeviceEvent);
#    else
    sendCoordinatesToPmeCudaMPI(sendPtr, sendSize, sendPmeCoordinatesFromGpu, coordinatesReadyOnDeviceEvent);
#    endif // GMX_THREAD_MPI

#else
    GMX_UNUSED_VALUE(sendPtr);
    GMX_UNUSED_VALUE(sendSize);
    GMX_UNUSED_VALUE(sendPmeCoordinatesFromGpu);
    GMX_UNUSED_VALUE(coordinatesReadyOnDeviceEvent);
#endif
}
void* PmePpCommGpu::Impl::getGpuForceStagingPtr()
{
    return static_cast<void*>(d_pmeForces_);
}

void* PmePpCommGpu::Impl::waitForcesReadyOrGetSynchronizer()
{
#if GMX_THREAD_MPI
    return static_cast<void*>(&forcesReadySynchronizer_);
#else
    MPI_Wait(&status_, MPI_STATUS_IGNORE);
    return nullptr;
#endif
}

PmePpCommGpu::PmePpCommGpu(MPI_Comm             comm,
                           int                  pmeRank,
                           const DeviceContext& deviceContext,
                           const DeviceStream&  deviceStream) :
    impl_(new Impl(comm, pmeRank, deviceContext, deviceStream))
{
}

PmePpCommGpu::~PmePpCommGpu() = default;

void PmePpCommGpu::reinit(int size)
{
    impl_->reinit(size);
}

void PmePpCommGpu::receiveForceFromPme(void* recvPtr, int recvSize, bool receivePmeForceToGpu)
{
    impl_->receiveForceFromPme(recvPtr, recvSize, receivePmeForceToGpu);
}

void PmePpCommGpu::sendCoordinatesToPme(void*                 sendPtr,
                                        int                   sendSize,
                                        bool                  sendPmeCoordinatesFromGpu,
                                        GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    impl_->sendCoordinatesToPme(sendPtr, sendSize, sendPmeCoordinatesFromGpu, coordinatesReadyOnDeviceEvent);
}

void* PmePpCommGpu::getGpuForceStagingPtr()
{
    return impl_->getGpuForceStagingPtr();
}

void* PmePpCommGpu::waitForcesReadyOrGetSynchronizer()
{
    return impl_->waitForcesReadyOrGetSynchronizer();
}

} // namespace gmx
