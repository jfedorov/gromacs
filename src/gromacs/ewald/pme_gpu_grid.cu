#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 * \brief Implements PME GPU halo exchange and PME GPU - Host FFT grid conversion
 * functions. These functions are used for PME decomposition in mixed-mode
 *
 * \author Gaurav Garg <gaugarg@nvidia.com>
 *
 * \ingroup module_ewald
 */

#include "gmxpre.h"

#include "pme_gpu_grid.h"

#include "config.h"

#include <cstdlib>

#include "gromacs/math/vec.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.cuh"

#include "pme.cuh"
#include "pme_gpu_types_host.h"
#include "pme_gpu_types.h"
#include "pme_gpu_types_host_impl.h"
#include "gromacs/fft/parallel_3dfft.h"

/*! \brief
 * A CUDA kernel which packs non-contiguous overlap data in Y-dimension
 *
 * \param[in] gm_realGrid          local grid
 * \param[in] gm_transferGrid      device array used to pack data
 * \param[in] offset               offset of y-overlap region
 * \param[in] overlapSize          overlap Size in y-overlap region
 * \param[in] pmeSize              Local PME grid size
 */
static __global__ void pmeGpuPackHaloY(const float* __restrict__ gm_realGrid,
                                       float* __restrict__ gm_transferGrid,
                                       int  offset,
                                       int  overlapSize,
                                       int3 pmeSize)
{
    int iz = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int ix = threadIdx.z + blockIdx.z * blockDim.z;

    // we might get iz greather than pmeSize.z when pmeSize.z is not
    // multiple of threadsAlongZDim(see below)
    if (iz >= pmeSize.z)
    {
        return;
    }

    int pmeIndex    = ix * pmeSize.y * pmeSize.z + (iy + offset) * pmeSize.z + iz;
    int packedIndex = ix * overlapSize * pmeSize.z + iy * pmeSize.z + iz;

    gm_transferGrid[packedIndex] = gm_realGrid[pmeIndex];
}

/*! \brief
 * A CUDA kernel which adds/puts grid overlap data received from neighboring rank in Y-dim
 *
 * \param[in] gm_realGrid          local grid
 * \param[in] gm_transferGrid      overlapping region from neighboring rank
 * \param[in] starty               offset of y-overlap region
 * \param[in] overlapSize          overlap Size in y-overlap region
 * \param[in] pmeSize              Local PME grid size
 */
template<bool forward>
static __global__ void pmeGpuAddHaloY(float* __restrict__ gm_realGrid,
                                      const float* __restrict__ gm_transferGrid,
                                      int  offset,
                                      int  overlapSize,
                                      int3 pmeSize)
{
    int iz = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int ix = threadIdx.z + blockIdx.z * blockDim.z;

    // we might get iz greather than pmeSize.z when pmeSize.z is not
    // multiple of threadsAlongZDim(see below)
    if (iz >= pmeSize.z)
    {
        return;
    }

    int pmeIndex    = ix * pmeSize.y * pmeSize.z + (iy + offset) * pmeSize.z + iz;
    int packedIndex = ix * overlapSize * pmeSize.z + iy * pmeSize.z + iz;

    if (forward)
    {
        gm_realGrid[pmeIndex] += gm_transferGrid[packedIndex];
    }
    else
    {
        gm_realGrid[pmeIndex] = gm_transferGrid[packedIndex];
    }
}

/*! \brief
 * A CUDA kernel which adds grid overlap data received from neighboring rank
 *
 * \param[in] gm_realGrid          local grid
 * \param[in] gm_transferGrid      overlapping region from neighboring rank
 * \param[in] size                 Number of elements in overlap region
 */
static __global__ void pmeGpuAddHalo(float* __restrict__ gm_realGrid,
                                     const float* __restrict__ gm_transferGrid,
                                     int size)
{
    int val = threadIdx.x + blockIdx.x * blockDim.x;
    if (val < size)
    {
        gm_realGrid[val] += gm_transferGrid[val];
    }
}

/*! \brief
 * A CUDA kernel which copies data from pme grid to FFT grid and back
 *
 * \param[in] gm_pmeGrid          local PME grid
 * \param[in] gm_fftGrid          local FFT grid
 * \param[in] fft_ndata           local FFT grid size without padding
 * \param[in] fft_size            local FFT grid padded size
 * \param[in] pme_size            local PME grid padded size
 */
template<bool forward>
static __global__ void pmegrid_to_fftgrid(float* __restrict__ gm_realGrid,
                                          float* __restrict__ gm_fftGrid,
                                          int3 fft_ndata,
                                          int3 fft_size,
                                          int3 pme_size)
{
    int iz = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int ix = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= fft_ndata.x || iy >= fft_ndata.y || iz >= fft_ndata.z)
    {
        return;
    }

    int fftidx   = ix * fft_size.y * fft_size.z + iy * fft_size.z + iz;
    int pmeIndex = ix * pme_size.y * pme_size.z + iy * pme_size.z + iz;

    if (forward)
    {
        gm_fftGrid[fftidx] = gm_realGrid[pmeIndex];
    }
    else
    {
        gm_realGrid[pmeIndex] = gm_fftGrid[fftidx];
    }
}

/*! \brief
 * Launches CUDA kernel to pack non-contiguous overlap data in Y-dimension
 *
 * \param[in]  pmeGpu              The PME GPU structure.
 * \param[in] overlapSize          overlap Size in y-overlap region
 * \param[in] yOffset              offset of y-overlap region
 * \param[in] localXSize           Local x size
 * \param[in] pmeSize              PME grid size
 * \param[in] realGrid             local grid
 * \param[in] packrdGrid           device array used to pack data
 */
static void packYData(const PmeGpu* pmeGpu,
                      int           overlapSize,
                      int           yOffset,
                      int           localXSize,
                      const ivec&   pmeSize,
                      float*        realGrid,
                      float*        packrdGrid)
{
    // keeping same as warp size for better coalescing
    // Not keeping to higher value such as 64 to avoid high masked out
    // inactive threads as FFT grid sizes tend to be quite small
    const int threadsAlongZDim = 32;

    // right grid
    KernelLaunchConfig config;
    config.blockSize[0]     = threadsAlongZDim;
    config.blockSize[1]     = overlapSize;
    config.blockSize[2]     = 1;
    config.gridSize[0]      = (pmeSize[ZZ] + threadsAlongZDim - 1) / threadsAlongZDim;
    config.gridSize[1]      = 1;
    config.gridSize[2]      = localXSize;
    config.sharedMemorySize = 0;


    auto kernelFn = pmeGpuPackHaloY;

    auto kernelArgs = prepareGpuKernelArguments(
            kernelFn, config, &realGrid, &packrdGrid, &yOffset, &overlapSize, &pmeSize);

    launchGpuKernel(kernelFn,
                    config,
                    pmeGpu->archSpecific->pmeStream_,
                    nullptr,
                    "PME Domdec GPU Pack Grid Halo Exchange",
                    kernelArgs);
}

/*! \brief
 * Launches CUDA kernel to reduce/unpack overlap data in Y-dimension
 *
 * \param[in]  pmeGpu              The PME GPU structure.
 * \param[in] overlapSize          overlap Size in y-overlap region
 * \param[in] yOffset              offset of y-overlap region
 * \param[in] localXSize           Local x size
 * \param[in] pmeSize              PME grid size
 * \param[in] realGrid             local grid
 * \param[in] packrdGrid           device array used to pack data
 */
template<bool forward>
static void reduceYData(const PmeGpu* pmeGpu,
                        int           overlapSize,
                        int           yOffset,
                        int           localXSize,
                        const ivec&   pmeSize,
                        float*        realGrid,
                        float*        packrdGrid)
{
    // keeping same as warp size for better coalescing
    // Not keeping to higher value such as 64 to avoid high masked out
    // inactive threads as FFT grid sizes tend to be quite small
    const int threadsAlongZDim = 32;

    // right grid
    KernelLaunchConfig config;
    config.blockSize[0]     = threadsAlongZDim;
    config.blockSize[1]     = overlapSize;
    config.blockSize[2]     = 1;
    config.gridSize[0]      = (pmeSize[ZZ] + threadsAlongZDim - 1) / threadsAlongZDim;
    config.gridSize[1]      = 1;
    config.gridSize[2]      = localXSize;
    config.sharedMemorySize = 0;

    auto kernelFn = pmeGpuAddHaloY<forward>;

    auto kernelArgs = prepareGpuKernelArguments(
            kernelFn, config, &realGrid, &packrdGrid, &yOffset, &overlapSize, &pmeSize);

    launchGpuKernel(kernelFn,
                    config,
                    pmeGpu->archSpecific->pmeStream_,
                    nullptr,
                    "PME Domdec GPU Pack Grid Halo Exchange",
                    kernelArgs);
}

/*! \brief
 * Launches CUDA kernel to reduce overlap data in X-dimension
 *
 * \param[in]  pmeGpu              The PME GPU structure.
 * \param[in] overlapSize          overlap Size in y-overlap region
 * \param[in] realGrid             local grid
 * \param[in] packrdGrid           device array used to pack data
 */
static void reduceXData(const PmeGpu* pmeGpu, int overlapSize, float* realGrid, float* packrdGrid)
{
    // launch reduction kernel
    const int threadsPerBlock = 64;

    KernelLaunchConfig config;
    config.blockSize[0]     = threadsPerBlock;
    config.blockSize[1]     = 1;
    config.blockSize[2]     = 1;
    config.gridSize[0]      = (overlapSize + threadsPerBlock - 1) / threadsPerBlock;
    config.gridSize[1]      = 1;
    config.gridSize[2]      = 1;
    config.sharedMemorySize = 0;

    auto kernelFn = pmeGpuAddHalo;

    auto kernelArgs = prepareGpuKernelArguments(kernelFn, config, &realGrid, &packrdGrid, &overlapSize);

    launchGpuKernel(kernelFn,
                    config,
                    pmeGpu->archSpecific->pmeStream_,
                    nullptr,
                    "PME Domdec GPU Apply Grid Halo Exchange",
                    kernelArgs);
}

void pmeGpuGridHaloExchange(const PmeGpu* pmeGpu)
{
    // Note here we are assuming that width of the chunks is not so small that we need to
    // transfer to/from multiple ranks i.e. that the distributed grid is at least order-1 points wide.

    auto* kernelParamsPtr = pmeGpu->kernelParams.get();
    ivec  local_pme_size;
    local_pme_size[XX] = kernelParamsPtr->grid.realGridSizePadded[XX];
    local_pme_size[YY] = kernelParamsPtr->grid.realGridSizePadded[YY];
    local_pme_size[ZZ] = kernelParamsPtr->grid.realGridSizePadded[ZZ];

    int extraGridLines = ceil(pmeGpu->common->rlist / pmeGpu->common->spacing);
    int overlapSize    = pmeGpu->common->pme_order - 1 + extraGridLines;

    // minor dimension
    if (pmeGpu->common->nnodes_minor > 1)
    {
        int rank  = pmeGpu->common->nodeid_minor;
        int size  = pmeGpu->common->nnodes_minor;
        int right = (rank + 1) % size;
        int left  = (rank + size - 1) % size;

        // Note that s2g0[size] is the grid size (array is allocated to size+1)
        int myGrid    = pmeGpu->common->s2g0y[rank + 1] - pmeGpu->common->s2g0y[rank];
        int rightGrid = pmeGpu->common->s2g0y[right + 1] - pmeGpu->common->s2g0y[right];
        int leftGrid  = pmeGpu->common->s2g0y[left + 1] - pmeGpu->common->s2g0y[left];

        // current implementation transfers from/to only immediate neighbours
        // in case overlap size is > slab width, we need to transfer data to multiple neighbours
        // Or, we should put a release assert which will NOT allow runs if overlapSize > slab size
        int overlapRecv  = std::min(overlapSize, myGrid);
        int overlapRight = std::min(overlapSize, rightGrid);
        int overlapLeft  = std::min(overlapSize, leftGrid);

        int pmegrid_nkx = pmeGpu->common->pmegrid_nk[XX];

        for (int gridIndex = 0; gridIndex < pmeGpu->common->ngrids; gridIndex++)
        {
            // launch packing kernel
            float* realGrid = pmeGpu->kernelParams->grid.d_realGrid[gridIndex];

            // Pack data that needs to be sent to right rank
            packYData(pmeGpu,
                      overlapRight,
                      myGrid,
                      pmegrid_nkx,
                      local_pme_size,
                      realGrid,
                      pmeGpu->archSpecific->d_sendGridRighty);

            // Pack data that needs to be sent to left rank
            packYData(pmeGpu,
                      overlapLeft,
                      local_pme_size[YY] - overlapLeft,
                      pmegrid_nkx,
                      local_pme_size,
                      realGrid,
                      pmeGpu->archSpecific->d_sendGridLefty);

            // synchronize before starting halo exchange
            pme_gpu_synchronize(pmeGpu);

            int        tag = 403; // Arbitrarily chosen
            MPI_Status status;

            // send data to right rank and recv from left rank
            MPI_Sendrecv(pmeGpu->archSpecific->d_sendGridRighty,
                         overlapRight * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         right,
                         tag,
                         pmeGpu->archSpecific->d_recvGridLefty,
                         overlapRecv * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         left,
                         tag,
                         pmeGpu->common->mpi_commy,
                         &status);

            // send data to left rank and recv from right rank
            MPI_Sendrecv(pmeGpu->archSpecific->d_sendGridLefty,
                         overlapLeft * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         left,
                         tag,
                         pmeGpu->archSpecific->d_recvGridRighty,
                         overlapRecv * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         right,
                         tag,
                         pmeGpu->common->mpi_commy,
                         &status);

            // reduce data received from left rank
            reduceYData<true>(pmeGpu,
                              overlapRecv,
                              0,
                              pmegrid_nkx,
                              local_pme_size,
                              realGrid,
                              pmeGpu->archSpecific->d_recvGridLefty);

            // reduce data received from right rank
            reduceYData<true>(pmeGpu,
                              overlapRecv,
                              myGrid - overlapRecv,
                              pmegrid_nkx,
                              local_pme_size,
                              realGrid,
                              pmeGpu->archSpecific->d_recvGridRighty);
        }
    }

    // major dimension
    if (pmeGpu->common->nnodes_major > 1)
    {
        int rank  = pmeGpu->common->nodeid_major;
        int size  = pmeGpu->common->nnodes_major;
        int right = (rank + 1) % size;
        int left  = (rank + size - 1) % size;

        // Note that s2g0[size] is the grid size (array is allocated to size+1)
        int myGrid    = pmeGpu->common->s2g0x[rank + 1] - pmeGpu->common->s2g0x[rank];
        int rightGrid = pmeGpu->common->s2g0x[right + 1] - pmeGpu->common->s2g0x[right];
        int leftGrid  = pmeGpu->common->s2g0x[left + 1] - pmeGpu->common->s2g0x[left];

        // current implementation transfers from/to only immediate neighbours
        // in case overlap size is > slab width, we need to transfer data to multiple neighbours
        // Or, we should put a release assert which will NOT allow runs if overlapSize > slab size
        int overlapRecv  = std::min(overlapSize, myGrid);
        int overlapRight = std::min(overlapSize, rightGrid);
        int overlapLeft  = std::min(overlapSize, leftGrid);

        int transferStartRight = myGrid * local_pme_size[YY] * local_pme_size[ZZ];
        int transferStartLeft =
                (local_pme_size[XX] - overlapLeft) * local_pme_size[YY] * local_pme_size[ZZ];

        // Current implementation transfers the whole grid along y, an optimization is
        // possible where only local y-length can be trasnferred
        // But, this will require executing packing kernel
        int transferSizeSendRight = overlapRight * local_pme_size[YY] * local_pme_size[ZZ];
        int transferSizeSendLeft  = overlapLeft * local_pme_size[YY] * local_pme_size[ZZ];
        int transferSizeRecv      = overlapRecv * local_pme_size[YY] * local_pme_size[ZZ];

        for (int gridIndex = 0; gridIndex < pmeGpu->common->ngrids; gridIndex++)
        {
            float* realGrid = pmeGpu->kernelParams->grid.d_realGrid[gridIndex];

            // synchronize before starting halo exchange
            pme_gpu_synchronize(pmeGpu);

            int tag = 403; // Arbitrarily chosen

            MPI_Status status;
            // send data to right rank and recv from left rank
            MPI_Sendrecv(&realGrid[transferStartRight],
                         transferSizeSendRight,
                         MPI_FLOAT,
                         right,
                         tag,
                         pmeGpu->archSpecific->d_recvGridLeftx,
                         transferSizeRecv,
                         MPI_FLOAT,
                         left,
                         tag,
                         pmeGpu->common->mpi_commx,
                         &status);

            // send data to left rank and recv from right rank
            MPI_Sendrecv(&realGrid[transferStartLeft],
                         transferSizeSendLeft,
                         MPI_FLOAT,
                         left,
                         tag,
                         pmeGpu->archSpecific->d_recvGridRightx,
                         transferSizeRecv,
                         MPI_FLOAT,
                         right,
                         tag,
                         pmeGpu->common->mpi_commx,
                         &status);

            // reduce data received from left rank
            reduceXData(pmeGpu, transferSizeRecv, realGrid, pmeGpu->archSpecific->d_recvGridLeftx);

            // reduce data received from right rank
            int    offset       = (myGrid - overlapRecv) * local_pme_size[YY] * local_pme_size[ZZ];
            float* offsetedGrid = realGrid + offset;
            reduceXData(pmeGpu, transferSizeRecv, offsetedGrid, pmeGpu->archSpecific->d_recvGridRightx);
        }
    }
}

void pmeGpuGridHaloExchangeReverse(const PmeGpu* pmeGpu)
{
    auto* kernelParamsPtr = pmeGpu->kernelParams.get();
    ivec  local_pme_size;
    local_pme_size[XX] = kernelParamsPtr->grid.realGridSizePadded[XX];
    local_pme_size[YY] = kernelParamsPtr->grid.realGridSizePadded[YY];
    local_pme_size[ZZ] = kernelParamsPtr->grid.realGridSizePadded[ZZ];

    int extraGridLines = ceil(pmeGpu->common->rlist / pmeGpu->common->spacing);
    int overlapSize    = pmeGpu->common->pme_order - 1 + extraGridLines;

    // minor dimension
    if (pmeGpu->common->nnodes_minor > 1)
    {
        int rank  = pmeGpu->common->nodeid_minor;
        int size  = pmeGpu->common->nnodes_minor;
        int right = (rank + 1) % size;
        int left  = (rank + size - 1) % size;

        int myGrid    = pmeGpu->common->s2g0y[rank + 1] - pmeGpu->common->s2g0y[rank];
        int rightGrid = pmeGpu->common->s2g0y[right + 1] - pmeGpu->common->s2g0y[right];
        int leftGrid  = pmeGpu->common->s2g0y[left + 1] - pmeGpu->common->s2g0y[left];

        // current implementation transfers from/to only immediate neighbours
        // in case overlap size is > slab width, we need to transfer data to multiple neighbours
        // Or, we should put a release assert which will NOT allow runs if overlapSize > slab size
        int overlapSend  = std::min(overlapSize, myGrid);
        int overlapRight = std::min(overlapSize, rightGrid);
        int overlapLeft  = std::min(overlapSize, leftGrid);

        int pmegrid_nkx = pmeGpu->common->pmegrid_nk[XX];

        for (int gridIndex = 0; gridIndex < pmeGpu->common->ngrids; gridIndex++)
        {
            // launch packing kernel
            float* realGrid = pmeGpu->kernelParams->grid.d_realGrid[gridIndex];

            // Pack data that needs to be sent to left rank
            packYData(pmeGpu,
                      overlapSend,
                      0,
                      pmegrid_nkx,
                      local_pme_size,
                      realGrid,
                      pmeGpu->archSpecific->d_sendGridLefty);

            // Pack data that needs to be sent to right rank
            packYData(pmeGpu,
                      overlapSend,
                      (myGrid - overlapSend),
                      pmegrid_nkx,
                      local_pme_size,
                      realGrid,
                      pmeGpu->archSpecific->d_sendGridRighty);

            // synchronize before starting halo exchange
            pme_gpu_synchronize(pmeGpu);

            int        tag = 403; // Arbitrarily chosen
            MPI_Status status;

            // send data to left rank and recv from right rank
            MPI_Sendrecv(pmeGpu->archSpecific->d_sendGridLefty,
                         overlapSend * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         left,
                         tag,
                         pmeGpu->archSpecific->d_recvGridRighty,
                         overlapRight * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         right,
                         tag,
                         pmeGpu->common->mpi_commy,
                         &status);

            // send data to right rank and recv from left rank
            MPI_Sendrecv(pmeGpu->archSpecific->d_sendGridRighty,
                         overlapSend * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         right,
                         tag,
                         pmeGpu->archSpecific->d_recvGridLefty,
                         overlapLeft * pmegrid_nkx * local_pme_size[ZZ],
                         MPI_FLOAT,
                         left,
                         tag,
                         pmeGpu->common->mpi_commy,
                         &status);

            // unpack data received from right rank
            reduceYData<false>(pmeGpu,
                               overlapRight,
                               myGrid,
                               pmegrid_nkx,
                               local_pme_size,
                               realGrid,
                               pmeGpu->archSpecific->d_recvGridRighty);

            // unpack data received from left rank
            reduceYData<false>(pmeGpu,
                               overlapLeft,
                               local_pme_size[YY] - overlapLeft,
                               pmegrid_nkx,
                               local_pme_size,
                               realGrid,
                               pmeGpu->archSpecific->d_recvGridLefty);
        }
    }

    // major dimension
    if (pmeGpu->common->nnodes_major > 1)
    {
        int rank  = pmeGpu->common->nodeid_major;
        int size  = pmeGpu->common->nnodes_major;
        int right = (rank + 1) % size;
        int left  = (rank + size - 1) % size;

        int myGrid    = pmeGpu->common->s2g0x[rank + 1] - pmeGpu->common->s2g0x[rank];
        int rightGrid = pmeGpu->common->s2g0x[right + 1] - pmeGpu->common->s2g0x[right];
        int leftGrid  = pmeGpu->common->s2g0x[left + 1] - pmeGpu->common->s2g0x[left];

        // current implementation transfers from/to only immediate neighbours
        // in case overlap size is > slab width, we need to transfer data to multiple neighbours
        // Or, we should put a release assert which will NOT allow runs if overlapSize > slab size
        int overlapSend  = std::min(overlapSize, myGrid);
        int overlapRight = std::min(overlapSize, rightGrid);
        int overlapLeft  = std::min(overlapSize, leftGrid);

        int transferstartRight = myGrid * local_pme_size[YY] * local_pme_size[ZZ];
        int transferstartLeft =
                (local_pme_size[XX] - overlapLeft) * local_pme_size[YY] * local_pme_size[ZZ];

        // Current implementation transfers the whole grid along y, an optimization is
        // possible where only local y-length can be trasnferred
        // But, this will require executing packing kernel
        int transferSizeSend      = overlapSend * local_pme_size[YY] * local_pme_size[ZZ];
        int transferSizeRecvRight = overlapRight * local_pme_size[YY] * local_pme_size[ZZ];
        int transferSizeRecvLeft  = overlapLeft * local_pme_size[YY] * local_pme_size[ZZ];

        for (int gridIndex = 0; gridIndex < pmeGpu->common->ngrids; gridIndex++)
        {
            float* realGrid = pmeGpu->kernelParams->grid.d_realGrid[gridIndex];

            pme_gpu_synchronize(pmeGpu);
            const int  tag = 403; // Arbitrarily chosen
            MPI_Status status;

            // send data to left rank and recv from right rank
            MPI_Sendrecv(&realGrid[0],
                         transferSizeSend,
                         MPI_FLOAT,
                         left,
                         tag,
                         &realGrid[transferstartRight],
                         transferSizeRecvRight,
                         MPI_FLOAT,
                         right,
                         tag,
                         pmeGpu->common->mpi_commx,
                         &status);

            // send data to right rank and recv from left rank
            int offset = (myGrid - overlapSend) * local_pme_size[YY] * local_pme_size[ZZ];
            MPI_Sendrecv(&realGrid[offset],
                         transferSizeSend,
                         MPI_FLOAT,
                         right,
                         tag,
                         &realGrid[transferstartLeft],
                         transferSizeRecvLeft,
                         MPI_FLOAT,
                         left,
                         tag,
                         pmeGpu->common->mpi_commx,
                         &status);
        }
    }
}

template<bool forward>
void convertPmeGridToFftGrid(const PmeGpu* pmeGpu, float* h_grid, gmx_parallel_3dfft_t* pfft_setup, const int gridIndex)
{
    ivec local_fft_ndata, local_fft_offset, local_fft_size;
    ivec local_pme_size;

    gmx_parallel_3dfft_real_limits(pfft_setup[gridIndex], local_fft_ndata, local_fft_offset, local_fft_size);

    local_pme_size[XX] = pmeGpu->kernelParams->grid.realGridSizePadded[XX];
    local_pme_size[YY] = pmeGpu->kernelParams->grid.realGridSizePadded[YY];
    local_pme_size[ZZ] = pmeGpu->kernelParams->grid.realGridSizePadded[ZZ];

    // this should be true in case of slab decomposition
    if (local_pme_size[ZZ] == local_fft_size[ZZ] && local_pme_size[YY] == local_fft_size[YY])
    {
        int fftSize = local_fft_size[ZZ] * local_fft_size[YY] * local_fft_ndata[XX];
        if (forward)
        {
            copyFromDeviceBuffer(h_grid,
                                 &pmeGpu->kernelParams->grid.d_realGrid[gridIndex],
                                 0,
                                 fftSize,
                                 pmeGpu->archSpecific->pmeStream_,
                                 pmeGpu->settings.transferKind,
                                 nullptr);
        }
        else
        {
            copyToDeviceBuffer(&pmeGpu->kernelParams->grid.d_realGrid[gridIndex],
                               h_grid,
                               0,
                               fftSize,
                               pmeGpu->archSpecific->pmeStream_,
                               pmeGpu->settings.transferKind,
                               nullptr);
        }
    }
    else
    {
        // launch copy kernel
        KernelLaunchConfig config;
        config.blockSize[0] = 32;
        config.blockSize[1] = 4;
        config.blockSize[2] = 1;
        config.gridSize[0]  = (local_fft_ndata[ZZ] + config.blockSize[0] - 1) / config.blockSize[0];
        config.gridSize[1]  = (local_fft_ndata[YY] + config.blockSize[1] - 1) / config.blockSize[1];
        config.gridSize[2]  = local_fft_ndata[XX];
        config.sharedMemorySize = 0;

        auto kernelFn = pmegrid_to_fftgrid<forward>;

        const auto kernelArgs =
                prepareGpuKernelArguments(kernelFn,
                                          config,
                                          &pmeGpu->kernelParams->grid.d_realGrid[gridIndex],
                                          &h_grid,
                                          &local_fft_ndata,
                                          &local_fft_size,
                                          &local_pme_size);

        launchGpuKernel(kernelFn,
                        config,
                        pmeGpu->archSpecific->pmeStream_,
                        nullptr,
                        "Convert PME grid to FFT grid",
                        kernelArgs);
    }

    if (forward)
    {
        pmeGpu->archSpecific->syncSpreadGridD2H.markEvent(pmeGpu->archSpecific->pmeStream_);
    }
}

template void convertPmeGridToFftGrid<true>(const PmeGpu*         pmeGpu,
                                            float*                h_grid,
                                            gmx_parallel_3dfft_t* pfft_setup,
                                            const int             gridIndex);

template void convertPmeGridToFftGrid<false>(const PmeGpu*         pmeGpu,
                                             float*                h_grid,
                                             gmx_parallel_3dfft_t* pfft_setup,
                                             const int             gridIndex);
