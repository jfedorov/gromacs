/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements class which recieves coordinates to GPU memory on PME task using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "gromacs/ewald/pme_pp_communication.h"
#include "pme_coordinate_receiver_gpu_impl.h"

#include "config.h"

#include "gromacs/ewald/pme_force_sender_gpu.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpueventsynchronizer.h"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

PmeCoordinateReceiverGpu::Impl::Impl(MPI_Comm               comm,
                                     const DeviceContext&   deviceContext,
                                     gmx::ArrayRef<PpRanks> ppRanks) :
    comm_(comm), ppRanks_(ppRanks), deviceContext_(deviceContext)
{
    request_.resize(ppRanks.size());
    ppSync_.resize(ppRanks.size());

    // Create streams to manage pipelining
    DeviceStream* stream;
    size_t        i = 0;
    while (i < ppRanks_.size())
    {
        stream = new DeviceStream(deviceContext_, DeviceStreamPriority::High, false);
        ppCommStream_.push_back(stream);
        i++;
    }
}

PmeCoordinateReceiverGpu::Impl::~Impl() = default;

void PmeCoordinateReceiverGpu::Impl::sendCoordinateBufferAddressToPpRanks(DeviceBuffer<RVec> d_x)
{
    // Need to send address to PP rank only for thread-MPI as PP rank pushes data using cudamemcpy
    if (GMX_THREAD_MPI)
    {
        int ind_start = 0;
        int ind_end   = 0;
        for (const auto& receiver : ppRanks_)
        {
            ind_start = ind_end;
            ind_end   = ind_start + receiver.numAtoms;

            // Data will be transferred directly from GPU.
            void* sendBuf = reinterpret_cast<void*>(&d_x[ind_start]);
#if GMX_MPI
            MPI_Send(&sendBuf, sizeof(void**), MPI_BYTE, receiver.rankId, 0, comm_);
#else
            GMX_UNUSED_VALUE(sendBuf);
#endif
        }
    }
}

/*! \brief Receive coordinate synchronizer pointer from the PP ranks. */
void PmeCoordinateReceiverGpu::Impl::receiveCoordinatesSynchronizerFromPpCudaDirect(int ppRank)
{
    GMX_ASSERT(GMX_THREAD_MPI,
               "receiveCoordinatesSynchronizerFromPpCudaDirect is expected to be called only for "
               "Thread-MPI");

    // Data will be pushed directly from PP task

#if GMX_MPI
    // Receive event from PP task
    // NOLINTNEXTLINE(bugprone-sizeof-expression)
    MPI_Irecv(&ppSync_[ppRank], sizeof(GpuEventSynchronizer*), MPI_BYTE, ppRank, 0, comm_, &request_[ppRank]);
#else
    GMX_UNUSED_VALUE(ppRank);
#endif
}

/*! \brief Receive coordinate data using CUDA-aware MPI */
void PmeCoordinateReceiverGpu::Impl::launchReceiveCoordinatesFromPpCudaMpi(DeviceBuffer<RVec> recvbuf,
                                                                           int numAtoms,
                                                                           int numBytes,
                                                                           int ppRank)
{
    GMX_ASSERT(GMX_LIB_MPI,
               "launchReceiveCoordinatesFromPpCudaMpi is expected to be called only for Lib-MPI");

#if GMX_MPI
    MPI_Irecv(&recvbuf[numAtoms], numBytes, MPI_BYTE, ppRank, eCommType_COORD_GPU, comm_, &request_[ppRank]);
#else
    GMX_UNUSED_VALUE(recvbuf);
    GMX_UNUSED_VALUE(numAtoms);
    GMX_UNUSED_VALUE(numBytes);
    GMX_UNUSED_VALUE(ppRank);
#endif
}

void PmeCoordinateReceiverGpu::Impl::synchronizeOnCoordinatesFromPpRanks(int senderIndex,
                                                                         const DeviceStream& deviceStream)
{
    // ensure PME calculation doesn't commence until coordinate data has been transferred
#if GMX_MPI
    MPI_Wait(&request_[senderIndex], MPI_STATUS_IGNORE);
    if (GMX_THREAD_MPI)
    {
        ppSync_[senderIndex]->enqueueWaitEvent(deviceStream);
    }
#endif
}

DeviceStream* PmeCoordinateReceiverGpu::Impl::ppCommStream(int senderIndex)
{
    return ppCommStream_[senderIndex];
}

int PmeCoordinateReceiverGpu::Impl::ppCommNumAtoms(int senderIndex)
{
    return ppRanks_[senderIndex].numAtoms;
}

int PmeCoordinateReceiverGpu::Impl::ppCommNumSenderRanks()
{
    return ppRanks_.size();
}

PmeCoordinateReceiverGpu::PmeCoordinateReceiverGpu(MPI_Comm               comm,
                                                   const DeviceContext&   deviceContext,
                                                   gmx::ArrayRef<PpRanks> ppRanks) :
    impl_(new Impl(comm, deviceContext, ppRanks))
{
}

PmeCoordinateReceiverGpu::~PmeCoordinateReceiverGpu() = default;

void PmeCoordinateReceiverGpu::sendCoordinateBufferAddressToPpRanks(DeviceBuffer<RVec> d_x)
{
    impl_->sendCoordinateBufferAddressToPpRanks(d_x);
}

void PmeCoordinateReceiverGpu::receiveCoordinatesSynchronizerFromPpCudaDirect(int ppRank)
{
    impl_->receiveCoordinatesSynchronizerFromPpCudaDirect(ppRank);
}

void PmeCoordinateReceiverGpu::launchReceiveCoordinatesFromPpCudaMpi(DeviceBuffer<RVec> recvbuf,
                                                                     int                numAtoms,
                                                                     int                numBytes,
                                                                     int                ppRank)
{
    impl_->launchReceiveCoordinatesFromPpCudaMpi(recvbuf, numAtoms, numBytes, ppRank);
}

void PmeCoordinateReceiverGpu::synchronizeOnCoordinatesFromPpRanks(int                 senderIndex,
                                                                   const DeviceStream& deviceStream)
{
    impl_->synchronizeOnCoordinatesFromPpRanks(senderIndex, deviceStream);
}

DeviceStream* PmeCoordinateReceiverGpu::ppCommStream(int senderIndex)
{
    return impl_->ppCommStream(senderIndex);
}

int PmeCoordinateReceiverGpu::ppCommNumAtoms(int senderIndex)
{
    return impl_->ppCommNumAtoms(senderIndex);
}

int PmeCoordinateReceiverGpu::ppCommNumSenderRanks()
{
    return impl_->ppCommNumSenderRanks();
}


} // namespace gmx
