#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016,2017,2018,2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements GPU 3D FFT routines for CUDA.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 *  \author Mark Abraham <mark.j.abraham@gmail.com>
 *  \ingroup module_fft
 */

#include "gmxpre.h"

#include "gpu_3dfft.h"

#include <hipfft/hipfft.h>

#include "gromacs/gpu_utils/device_stream.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/devicebuffer.cuh"
#include "gromacs/ewald/pme_gpu_types_host.h"
#include "gromacs/ewald/pme_gpu_internal.h"
#include "gromacs/ewald/pme.cuh"

#define UCX_MPIALLTOALLV_BUG_HACK 1

namespace gmx
{

class Gpu3dFft::Impl
{
public:
    Impl(const PmeGpu*        pmeGpu,
         ivec                 realGridSize,
         ivec                 realGridSizePadded,
         ivec                 complexGridSize,
         ivec                 complexGridSizePadded,
         bool                 useDecomposition,
         bool                 performOutOfPlaceFFT,
         const DeviceContext& context,
         const DeviceStream&  pmeStream,
         DeviceBuffer<float>  realGrid,
         DeviceBuffer<float>  complexGrid,
         DeviceBuffer<float>  complexGrid2);
    ~Impl();

    hipfftHandle   planR2C_;
    hipfftHandle   planC2R_;
    hipfftReal*    realGrid_;
    hipfftComplex* complexGrid_;
    hipfftComplex* complexGrid2_;
    ivec complexGridSizePadded_;

    const PmeGpu* pmeGpu_;

    /*! \brief
     * CUDA stream used for PME computation
     */
    const DeviceStream& stream_;

    /*! \brief
     * 2D and 1D cufft plans used for distributed fft implementation
     */
    hipfftHandle planR2C2D_;
    hipfftHandle planC2R2D_;
    hipfftHandle planC2C1D_;

    /*! \brief
     * MPI complex type
     */
    MPI_Datatype complexType_;

    /*! \brief
     * MPI communicator for PME ranks
     */
    MPI_Comm mpi_comm_;

    /*! \brief
     * total ranks within PME group
     */
    int mpiSize_;

    /*! \brief
     * current local mpi rank within PME group
     */
    int mpiRank_;

    /*! \brief
     * Max local grid size in X-dim (used during transposes in forward pass)
     */
    int xMax_;

    /*! \brief
     * Max local grid size in Y-dim (used during transposes in reverse pass)
     */
    int yMax_;

    /*! \brief
     * device array containing 1D decomposition size in X-dim (forwarad pass)
     */
    DeviceBuffer<int> d_xBlockSizes_;

    /*! \brief
     * device array containing 1D decomposition size in Y-dim (reverse pass)
     */
    DeviceBuffer<int> d_yBlockSizes_;

    /*! \brief
     * device arrays for local interpolation grid start values in X-dim
     * (used during transposes in forward pass)
     */
    DeviceBuffer<int> d_s2g0x_;

    /*! \brief
     * device arrays for local interpolation grid start values in Y-dim
     * (used during transposes in reverse pass)
     */
    DeviceBuffer<int> d_s2g0y_;

    /*! \brief
     * host array containing 1D decomposition size in X-dim (forwarad pass)
     */
    gmx::HostVector<int> h_xBlockSizes_;

    /*! \brief
     * host array containing 1D decomposition size in Y-dim (reverse pass)
     */
    gmx::HostVector<int> h_yBlockSizes_;

    /*! \brief
     * host array for local interpolation grid start values in Y-dim
     */
    gmx::HostVector<int> h_s2g0y_;

    /*! \brief
     * device array big enough to hold grid overlapping region
     * used during grid halo exchange
     */
    DeviceBuffer<float> d_transferGrid_;

    /*! \brief
     * count and displacement arrays used in MPI_Alltoall call
     *
     */
    int *sendCount_, *sendDisp_;
    int *recvCount_, *recvDisp_;

#        if UCX_MPIALLTOALLV_BUG_HACK
    /*! \brief
     * count arrays used in MPI_Alltoall call which has no self copies
     *
     */
    int *sendCountTemp_, *recvCountTemp_;
#        endif
};

static void handleCufftError(hipfftResult_t status, const char* msg)
{
    if (status != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "%s (error code %d)\n", msg, status);
    }
}

// CUDA block size x and y-dim
constexpr int c_threads = 16;

/*! \brief
 * A CUDA kernel which converts grid from XYZ to YZX layout in case of forward fft
 * and converts from YZX to XYZ layout in case of reverse fft
 *
 * \tparam[in] forward            Forward pass or reverse pass
 *
 * \param[in] gm_arrayIn          Input local grid
 * \param[in] gm_arrayOut         Output local grid in converted layout
 * \param[in] sizeX               Grid size in X-dim.
 * \param[in] sizeY               Grid size in Y-dim.
 * \param[in] sizeZ               Grid size in Z-dim.
 */
template<bool forward>
static __global__ void transposeXyzToYzxKernel(const hipfftComplex* __restrict__ gm_arrayIn,
                                               hipfftComplex* __restrict__ gm_arrayOut,
                                               const int sizeX,
                                               const int sizeY,
                                               const int sizeZ)
{
    __shared__ hipfftComplex sm_temp[c_threads][c_threads];
    int                     x = blockIdx.x * blockDim.x + threadIdx.x;
    int                     y = blockIdx.y;
    int                     z = blockIdx.z * blockDim.z + threadIdx.z;

    // use threads in other order for xyz (works as blockDim.x == blockDim.z)
    int xt = blockIdx.x * blockDim.x + threadIdx.z;
    int zt = blockIdx.z * blockDim.z + threadIdx.x;

    int  xyzIndex = zt + y * sizeZ + xt * sizeY * sizeZ;
    int  yzxIndex = x + z * sizeX + y * sizeX * sizeZ;
    int  inIndex, outIndex;
    bool validIn, validOut;

    if (forward) // xyz to yzx
    {
        inIndex  = xyzIndex;
        outIndex = yzxIndex;
        validIn  = (xt < sizeX && zt < sizeZ);
        validOut = (x < sizeX && z < sizeZ);
    }
    else // yzx to xyz
    {
        inIndex  = yzxIndex;
        outIndex = xyzIndex;
        validIn  = (x < sizeX && z < sizeZ);
        validOut = (xt < sizeX && zt < sizeZ);
    }

    if (validIn)
    {
        sm_temp[threadIdx.x][threadIdx.z] = gm_arrayIn[inIndex];
    }
    __syncthreads();

    if (validOut)
    {
        gm_arrayOut[outIndex] = sm_temp[threadIdx.z][threadIdx.x];
    }
}

/*! \brief
 * A CUDA kernel which merges multiple blocks in YZX layout from different ranks
 *
 * \param[in] gm_arrayIn          Input local grid
 * \param[in] gm_arrayOut         Output local grid in converted layout
 * \param[in] sizeX               Grid size in X-dim.
 * \param[in] sizeY               Grid size in Y-dim.
 * \param[in] sizeZ               Grid size in Z-dim.
 * \param[in] xBlockSizes         Array containing X-block sizes for each rank
 * \param[in] xOffset             Array containing grid offsets for each rank
 */
static __global__ void convertBlockedYzxToYzxKernel(const hipfftComplex* __restrict__ gm_arrayIn,
                                                    hipfftComplex* __restrict__ gm_arrayOut,
                                                    const int sizeX,
                                                    const int sizeY,
                                                    const int sizeZ,
                                                    const int* __restrict__ xBlockSizes,
                                                    const int* __restrict__ xOffset)
{
    // no need to cache block unless x_block_size is small
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    int region = blockIdx.z;
    int xLocal = thread % xBlockSizes[region];
    int z      = thread / xBlockSizes[region];
    int y      = blockIdx.y;
    int x      = xOffset[region] + xLocal;

    int indexIn  = xLocal + xBlockSizes[region] * (z + sizeZ * y) + xOffset[region] * sizeY * sizeZ;
    int indexOut = x + sizeX * (z + sizeZ * y);

    if (x < xOffset[region + 1] && z < sizeZ)
    {
        gm_arrayOut[indexOut] = gm_arrayIn[indexIn];
    }
}

/*! \brief
 * A CUDA kernel which merges multiple blocks in XYZ layout from different ranks
 *
 * \param[in] gm_arrayIn          Input local grid
 * \param[in] gm_arrayOut         Output local grid in converted layout
 * \param[in] sizeX               Grid size in X-dim.
 * \param[in] sizeY               Grid size in Y-dim.
 * \param[in] sizeZ               Grid size in Z-dim.
 * \param[in] yBlockSizes         Array containing Y-block sizes for each rank
 * \param[in] yOffset             Array containing grid offsets for each rank
 */
static __global__ void convertBlockedXyzToXyzKernel(const hipfftComplex* __restrict__ gm_arrayIn,
                                                    hipfftComplex* __restrict__ gm_arrayOut,
                                                    const int sizeX,
                                                    const int sizeY,
                                                    const int sizeZ,
                                                    const int* __restrict__ yBlockSizes,
                                                    const int* __restrict__ yOffset)
{
    int x      = blockIdx.y;
    int yz     = blockIdx.x * blockDim.x + threadIdx.x;
    int region = blockIdx.z;
    int z      = yz % sizeZ;
    int yLocal = yz / sizeZ;

    int y        = yLocal + yOffset[region];
    int indexIn  = z + sizeZ * (yLocal + yBlockSizes[region] * x + sizeX * yOffset[region]);
    int indexOut = z + sizeZ * (y + sizeY * x);

    if (y < yOffset[region + 1] && z < sizeZ)
    {
        gm_arrayOut[indexOut] = gm_arrayIn[indexIn];
    }
}

template<bool forward>
static void transposeXyzToYzx(hipfftComplex* arrayIn, hipfftComplex* arrayOut, int sizeX, int sizeY, int sizeZ, const DeviceStream& stream)
{
    KernelLaunchConfig config;
    config.blockSize[0]     = c_threads;
    config.blockSize[1]     = 1;
    config.blockSize[2]     = c_threads;
    config.gridSize[0]      = (sizeX + c_threads - 1) / c_threads;
    config.gridSize[1]      = sizeY;
    config.gridSize[2]      = (sizeZ + c_threads - 1) / c_threads;
    config.sharedMemorySize = 0;


    auto kernelFn = transposeXyzToYzxKernel<forward>;

    const auto kernelArgs =
            prepareGpuKernelArguments(kernelFn, config, &arrayIn, &arrayOut, &sizeX, &sizeY, &sizeZ);

    launchGpuKernel(kernelFn, config, stream, nullptr, "PME FFT GPU grid transpose", kernelArgs);
}

static void convertBlockedYzxToYzx(hipfftComplex* arrayIn,
                                              hipfftComplex* arrayOut,
                                              int           sizeX,
                                              int           sizeY,
                                              int           sizeZ,
                                              int*          xBlockSizes,
                                              int*          xOffsets,
                                              int           numRegions,
                                              int           maxRegionSize,
                                              const DeviceStream& stream)
{
    int blockDim = c_threads * c_threads;
    int sizexz   = maxRegionSize * sizeZ;

    KernelLaunchConfig config;
    config.blockSize[0]     = blockDim;
    config.blockSize[1]     = 1;
    config.blockSize[2]     = 1;
    config.gridSize[0]      = (sizexz + blockDim - 1) / blockDim;
    config.gridSize[1]      = sizeY;
    config.gridSize[2]      = numRegions;
    config.sharedMemorySize = 0;


    auto kernelFn = convertBlockedYzxToYzxKernel;

    const auto kernelArgs = prepareGpuKernelArguments(
            kernelFn, config, &arrayIn, &arrayOut, &sizeX, &sizeY, &sizeZ, &xBlockSizes, &xOffsets);

    launchGpuKernel(kernelFn, config, stream, nullptr, "PME FFT GPU grid rearrange", kernelArgs);
}

static void convertBlockedXyzToXyz(hipfftComplex* arrayIn,
                                    hipfftComplex* arrayOut,
                                    int           sizeX,
                                    int           sizeY,
                                    int           sizeZ,
                                    int*          yBlockSizes,
                                    int*          yOffsets,
                                    int           numRegions,
                                    int           maxRegionSize,
                                    const DeviceStream& stream)
{
    int blockDim = c_threads * c_threads;
    int sizexz   = maxRegionSize * sizeZ;

    KernelLaunchConfig config;
    config.blockSize[0]     = blockDim;
    config.blockSize[1]     = 1;
    config.blockSize[2]     = 1;
    config.gridSize[0]      = (sizexz + blockDim - 1) / blockDim;
    config.gridSize[1]      = sizeX;
    config.gridSize[2]      = numRegions;
    config.sharedMemorySize = 0;


    auto kernelFn = convertBlockedXyzToXyzKernel;

    const auto kernelArgs = prepareGpuKernelArguments(
            kernelFn, config, &arrayIn, &arrayOut, &sizeX, &sizeY, &sizeZ, &yBlockSizes, &yOffsets);

    launchGpuKernel(kernelFn, config, stream, nullptr, "PME FFT GPU grid rearrange", kernelArgs);
}

Gpu3dFft::Impl::Impl(const PmeGpu* pmeGpu,
                     ivec       realGridSize,
                     ivec       realGridSizePadded,
                     ivec       complexGridSize,
                     ivec       complexGridSizePadded,
                     const bool useDecomposition,
                     const bool /*performOutOfPlaceFFT*/,
                     const DeviceContext& context,
                     const DeviceStream& pmeStream,
                     DeviceBuffer<float> realGrid,
                     DeviceBuffer<float> complexGrid,
                     DeviceBuffer<float> complexGrid2) :
    realGrid_(reinterpret_cast<hipfftReal*>(realGrid)),
    complexGrid_(reinterpret_cast<hipfftComplex*>(complexGrid)),
    complexGrid2_(reinterpret_cast<hipfftComplex*>(complexGrid2)),
    pmeGpu_(pmeGpu),
    stream_(pmeStream)
{
    for (int i = 0; i < DIM; i++)
    {
        complexGridSizePadded_[i] = complexGridSizePadded[i];
    }
    const int complexGridSizePaddedTotal =
            complexGridSizePadded[XX] * complexGridSizePadded[YY] * complexGridSizePadded[ZZ];
    const int realGridSizePaddedTotal =
            realGridSizePadded[XX] * realGridSizePadded[YY] * realGridSizePadded[ZZ];

    realGrid_ = realGrid;

    GMX_RELEASE_ASSERT(realGrid_, "Bad (null) input real-space grid");
    GMX_RELEASE_ASSERT(complexGrid_, "Bad (null) input complex grid");


    hipfftResult_t result;
    /* Commented code for a simple 3D grid with no padding */
    /*
       result = hipfftPlan3d(&planR2C_, realGridSize[XX], realGridSize[YY], realGridSize[ZZ],
       HIPFFT_R2C); handleCufftError(result, "hipfftPlan3d R2C plan failure");

       result = hipfftPlan3d(&planC2R_, realGridSize[XX], realGridSize[YY], realGridSize[ZZ],
       HIPFFT_C2R); handleCufftError(result, "hipfftPlan3d C2R plan failure");
     */

    hipStream_t stream = pmeStream.stream();
    GMX_RELEASE_ASSERT(stream, "Can not use the default CUDA stream for PME cuFFT");

    if (!useDecomposition)
    {
        int rank  = 3;
        int batch = 1;
        result    = hipfftPlanMany(&planR2C_,
                               rank,
                               realGridSize,
                               realGridSizePadded,
                               1,
                               realGridSizePaddedTotal,
                               complexGridSizePadded,
                               1,
                               complexGridSizePaddedTotal,
                               HIPFFT_R2C,
                               batch);
        handleCufftError(result, "hipfftPlanMany R2C plan failure");
        result = hipfftSetStream(planR2C_, stream);
        handleCufftError(result, "hipfftSetStream R2C failure");


        result = hipfftPlanMany(&planC2R_,
                               rank,
                               realGridSize,
                               complexGridSizePadded,
                               1,
                               complexGridSizePaddedTotal,
                               realGridSizePadded,
                               1,
                               realGridSizePaddedTotal,
                               HIPFFT_C2R,
                               batch);
        handleCufftError(result, "hipfftPlanMany C2R plan failure");
        result = hipfftSetStream(planC2R_, stream);
        handleCufftError(result, "hipfftSetStream C2R failure");
    }

    int mpiSize   = 1;
    int mpiRank   = 0;

    // count and displacement arrays used in MPI_Alltoall call
    sendCount_ = sendDisp_ = recvCount_ = recvDisp_ = NULL;
#    if UCX_MPIALLTOALLV_BUG_HACK
    sendCountTemp_ = recvCountTemp_ = NULL;
#    endif

    // local grid size along decmposed dimension
    d_xBlockSizes_ = d_yBlockSizes_ = NULL;

    // device arrays keeping local grid offsets
    d_s2g0x_ = d_s2g0y_ = NULL;

    // device memory to transfer overlapping regions between ranks
    d_transferGrid_ = NULL;
    if (useDecomposition)
    {
        changePinningPolicy(&h_xBlockSizes_, gmx::PinningPolicy::PinnedIfSupported);
        changePinningPolicy(&h_yBlockSizes_, gmx::PinningPolicy::PinnedIfSupported);
        changePinningPolicy(&h_s2g0y_, gmx::PinningPolicy::PinnedIfSupported);

        const int complexGridSizePaddedTotal2D = complexGridSizePadded[YY] * complexGridSizePadded[ZZ];
        const int realGridSizePaddedTotal2D    = realGridSizePadded[YY] * realGridSizePadded[ZZ];

        int localx = realGridSize[XX];
        int localy = realGridSize[YY];

        MPI_Comm_size(pmeGpu->common->mpi_commx, &mpiSize);
        MPI_Comm_rank(pmeGpu->common->mpi_commx, &mpiRank);
        mpi_comm_  = pmeGpu->common->mpi_commx;
        sendCount_ = (int*)malloc(mpiSize * sizeof(int));
        sendDisp_  = (int*)malloc(mpiSize * sizeof(int));
        recvCount_ = (int*)malloc(mpiSize * sizeof(int));
        recvDisp_  = (int*)malloc(mpiSize * sizeof(int));
        h_xBlockSizes_.resize(mpiSize);
        h_yBlockSizes_.resize(mpiSize);
        h_s2g0y_.resize(mpiSize + 1);
        allocateDeviceBuffer(&d_xBlockSizes_, mpiSize, context);
        allocateDeviceBuffer(&d_yBlockSizes_, mpiSize, context);
        allocateDeviceBuffer(&d_s2g0x_, (mpiSize + 1), context);
        allocateDeviceBuffer(&d_s2g0y_, (mpiSize + 1), context);

        localx = pmeGpu_->common->s2g0x[mpiRank + 1] - pmeGpu_->common->s2g0x[mpiRank];

        for (int i = 0; i < mpiSize; i++)
        {
            h_s2g0y_[i] = (i * complexGridSizePadded[YY] + 0) / mpiSize;
        }
        h_s2g0y_[mpiSize] = complexGridSizePadded[YY];

        localy        = h_s2g0y_[mpiRank + 1] - h_s2g0y_[mpiRank];
        int totalSend = 0;
        int totalRecv = 0;
        int xmax      = 0;
        int ymax      = 0;
        for (int i = 0; i < mpiSize; i++)
        {
            int ix            = pmeGpu_->common->s2g0x[i + 1] - pmeGpu_->common->s2g0x[i];
            int iy            = h_s2g0y_[i + 1] - h_s2g0y_[i];
            h_xBlockSizes_[i] = ix;
            h_yBlockSizes_[i] = iy;
            if (xmax < ix)
                xmax = ix;
            if (ymax < iy)
                ymax = iy;
            sendCount_[i] = complexGridSize[ZZ] * localx * iy;
            recvCount_[i] = complexGridSize[ZZ] * localy * ix;
            sendDisp_[i]  = totalSend;
            recvDisp_[i]  = totalRecv;
            totalSend += sendCount_[i];
            totalRecv += recvCount_[i];
        }
        xMax_ = xmax;
        yMax_ = ymax;
        copyToDeviceBuffer(
                &d_s2g0x_, pmeGpu_->common->s2g0x.data(), 0, (mpiSize + 1), stream_, GpuApiCallBehavior::Sync, nullptr);
        copyToDeviceBuffer(
                &d_xBlockSizes_, h_xBlockSizes_.data(), 0, mpiSize, stream_, GpuApiCallBehavior::Async, nullptr);
        copyToDeviceBuffer(
                &d_yBlockSizes_, h_yBlockSizes_.data(), 0, mpiSize, stream_, GpuApiCallBehavior::Async, nullptr);
        copyToDeviceBuffer(
                &d_s2g0y_, h_s2g0y_.data(), 0, (mpiSize + 1), stream_, GpuApiCallBehavior::Async, nullptr);

        allocateDeviceBuffer(
                &d_transferGrid_, xmax * realGridSizePadded[YY] * realGridSizePadded[ZZ], context);

#    if UCX_MPIALLTOALLV_BUG_HACK
        sendCountTemp_ = (int*)malloc(mpiSize * sizeof(int));
        recvCountTemp_ = (int*)malloc(mpiSize * sizeof(int));

        memcpy(sendCountTemp_, sendCount_, mpiSize * sizeof(int));
        memcpy(recvCountTemp_, recvCount_, mpiSize * sizeof(int));

        // don't make any self copies. UCX has perf issues with self copies
        sendCountTemp_[mpiRank] = 0;
        recvCountTemp_[mpiRank] = 0;
#    endif

        int rank  = 2;
        int batch = localx;
        // split 3d fft as 2D fft and 1d fft to implement distributed fft
        result = hipfftPlanMany(&planR2C2D_,
                               rank,
                               &realGridSize[YY],
                               &realGridSizePadded[YY],
                               1,
                               realGridSizePaddedTotal2D,
                               &complexGridSizePadded[YY],
                               1,
                               complexGridSizePaddedTotal2D,
                               HIPFFT_R2C,
                               batch);
        handleCufftError(result, "hipfftPlanMany 2D R2C plan failure");
        result = hipfftSetStream(planR2C2D_, stream);
        handleCufftError(result, "hipfftSetStream R2C failure");

        result = hipfftPlanMany(&planC2R2D_,
                               rank,
                               &realGridSize[YY],
                               &complexGridSizePadded[YY],
                               1,
                               complexGridSizePaddedTotal2D,
                               &realGridSizePadded[YY],
                               1,
                               realGridSizePaddedTotal2D,
                               HIPFFT_C2R,
                               batch);
        handleCufftError(result, "hipfftPlanMany 2D C2R plan failure");
        result = hipfftSetStream(planC2R2D_, stream);
        handleCufftError(result, "hipfftSetStream C2R failure");

        rank   = 1;
        batch  = localy * complexGridSize[ZZ];
        result = hipfftPlanMany(&planC2C1D_,
                               rank,
                               &complexGridSize[XX], // 1D C2C part of the R2C
                               &complexGridSizePadded[XX],
                               1,
                               complexGridSizePadded[XX],
                               &complexGridSizePadded[XX],
                               1,
                               complexGridSizePadded[XX],
                               HIPFFT_C2C,
                               batch);
        handleCufftError(result, "hipfftPlanMany  1D C2C plan failure");
        result = hipfftSetStream(planC2C1D_, stream);
        handleCufftError(result, "hipfftSetStream C2C failure");

        MPI_Type_contiguous(2, MPI_FLOAT, &complexType_);
        MPI_Type_commit(&complexType_);
    }
    mpiSize_ = mpiSize;
    mpiRank_ = mpiRank;
}

Gpu3dFft::Impl::~Impl()
{
    hipfftResult_t result;
    if (!pme_gpu_settings(pmeGpu_).useDecomposition)
    {
        result = hipfftDestroy(planR2C_);
        handleCufftError(result, "hipfftDestroy R2C failure");
        result = hipfftDestroy(planC2R_);
        handleCufftError(result, "hipfftDestroy C2R failure");
    }
    else
    {
        result = hipfftDestroy(planR2C2D_);
        handleCufftError(result, "hipfftDestroy R2C failure");
        result = hipfftDestroy(planC2R2D_);
        handleCufftError(result, "hipfftDestroy C2R failure");
        result = hipfftDestroy(planC2C1D_);
        handleCufftError(result, "hipfftDestroy C2C failure");

        MPI_Type_free(&complexType_);

        free(sendCount_);
        free(sendDisp_);
        free(recvCount_);
        free(recvDisp_);
        freeDeviceBuffer(&d_xBlockSizes_);
        freeDeviceBuffer(&d_yBlockSizes_);
        freeDeviceBuffer(&d_s2g0x_);
        freeDeviceBuffer(&d_s2g0y_);

#    if UCX_MPIALLTOALLV_BUG_HACK
        free(sendCountTemp_);
        free(recvCountTemp_);

#    endif // UCX_MPIALLTOALLV_BUG_HACK
    }
}

void Gpu3dFft::perform3dFft(gmx_fft_direction dir, CommandEvent* /*timingEvent*/)
{
    hipfftResult_t result;
    if (!pme_gpu_settings(impl_->pmeGpu_).useDecomposition)
    {
        if (dir == GMX_FFT_REAL_TO_COMPLEX)
        {
            result = hipfftExecR2C(impl_->planR2C_, impl_->realGrid_, impl_->complexGrid_);
            handleCufftError(result, "cuFFT R2C execution failure");
        }
        else
        {
            result = hipfftExecC2R(impl_->planC2R_, impl_->complexGrid_, impl_->realGrid_);
            handleCufftError(result, "cuFFT C2R execution failure");
        }
    }
    else
    {
        int                           localx, localy;
        localx = impl_->pmeGpu_->common->s2g0x[impl_->mpiRank_ + 1] - impl_->pmeGpu_->common->s2g0x[impl_->mpiRank_];
        localy = impl_->h_s2g0y_[impl_->mpiRank_ + 1] - impl_->h_s2g0y_[impl_->mpiRank_];

        if (dir == GMX_FFT_REAL_TO_COMPLEX)
        {
            // 2D FFT
            result = hipfftExecR2C(impl_->planR2C2D_, impl_->realGrid_, impl_->complexGrid_);
            handleCufftError(result, "cuFFT R2C 2D execution failure");
            // Transpose and communicate
            transposeXyzToYzx<true>(
                    impl_->complexGrid_, impl_->complexGrid2_, localx, impl_->complexGridSizePadded_[YY], impl_->complexGridSizePadded_[ZZ], impl_->stream_);
            pme_gpu_synchronize(impl_->pmeGpu_);

#    if UCX_MPIALLTOALLV_BUG_HACK

            // self copy on the same rank
            hipMemcpyAsync(impl_->complexGrid_ + impl_->recvDisp_[impl_->mpiRank_],
                            impl_->complexGrid2_ + impl_->sendDisp_[impl_->mpiRank_],
                            impl_->recvCount_[impl_->mpiRank_] * sizeof(hipfftComplex),
                            hipMemcpyDeviceToDevice,
                            impl_->stream_.stream());

            // copy to other ranks. UCX has perf issues if self copies are made in MPI_Alltoallv call
            MPI_Alltoallv(impl_->complexGrid2_,
                          impl_->sendCountTemp_,
                          impl_->sendDisp_,
                          impl_->complexType_,
                          impl_->complexGrid_,
                          impl_->recvCountTemp_,
                          impl_->recvDisp_,
                          impl_->complexType_,
                          impl_->mpi_comm_);

#    else
            // MPI_Alltoallv has perf issues where copy to self is too slow. above implementation takes care of that
            MPI_Alltoallv(impl_->complexGrid2_,
                          impl_->sendCount_,
                          impl_->sendDisp_,
                          impl_->complexType_,
                          impl_->complexGrid_,
                          impl_->recvCount_,
                          impl_->recvDisp_,
                          impl_->complexType_,
                          impl_->mpi_comm_);
#    endif

            // make data in proper layout once different blocks are received from different MPI ranks
            convertBlockedYzxToYzx(impl_->complexGrid_,
                                   impl_->complexGrid2_,
                                   impl_->complexGridSizePadded_[XX],
                                   localy,
                                   impl_->complexGridSizePadded_[ZZ],
                                   impl_->d_xBlockSizes_,
                                   impl_->d_s2g0x_,
                                   impl_->mpiSize_,
                                   impl_->xMax_,
                                   impl_->stream_);
            // 1D FFT
            result = hipfftExecC2C(impl_->planC2C1D_, impl_->complexGrid2_, impl_->complexGrid_, HIPFFT_FORWARD);
            handleCufftError(result, "cuFFT C2C 1D execution failure");
        }
        else
        {
            // 1D FFT
            result = hipfftExecC2C(impl_->planC2C1D_, impl_->complexGrid_, impl_->complexGrid2_, HIPFFT_BACKWARD);
            handleCufftError(result, "cuFFT C2C 1D execution failure");
            // transpose and communicate
            transposeXyzToYzx<false>(
                    impl_->complexGrid2_, impl_->complexGrid_, impl_->complexGridSizePadded_[XX], localy, impl_->complexGridSizePadded_[ZZ], impl_->stream_);
            pme_gpu_synchronize(impl_->pmeGpu_);

#    if UCX_MPIALLTOALLV_BUG_HACK
            // self copy on the same rank
            hipMemcpyAsync(impl_->complexGrid2_ + impl_->recvDisp_[impl_->mpiRank_],
                            impl_->complexGrid_ + impl_->sendDisp_[impl_->mpiRank_],
                            impl_->recvCount_[impl_->mpiRank_] * sizeof(hipfftComplex),
                            hipMemcpyDeviceToDevice,
                            impl_->stream_.stream());

            // copy to other ranks. UCX has perf issues if self copies are made in MPI_Alltoallv call
            MPI_Alltoallv(impl_->complexGrid_,
                          impl_->sendCountTemp_,
                          impl_->sendDisp_,
                          impl_->complexType_,
                          impl_->complexGrid2_,
                          impl_->recvCountTemp_,
                          impl_->recvDisp_,
                          impl_->complexType_,
                          impl_->mpi_comm_);

#    else
            MPI_Alltoallv(impl_->complexGrid_,
                          impl_->sendCount_,
                          impl_->sendDisp_,
                          impl_->complexType_,
                          impl_->complexGrid2_,
                          impl_->recvCount_,
                          impl_->recvDisp_,
                          impl_->complexType_,
                          impl_->mpi_comm_);
#    endif

            // make data in proper layout once different blocks are received from different MPI ranks
            convertBlockedXyzToXyz(impl_->complexGrid2_,
                                   impl_->complexGrid_,
                                   localx,
                                   impl_->complexGridSizePadded_[YY],
                                   impl_->complexGridSizePadded_[ZZ],
                                   impl_->d_yBlockSizes_,
                                   impl_->d_s2g0y_,
                                   impl_->mpiSize_,
                                   impl_->yMax_,
                                   impl_->stream_);
            // 2D
            result = hipfftExecC2R(impl_->planC2R2D_, impl_->complexGrid_, impl_->realGrid_);
            handleCufftError(result, "cuFFT C2R 2D execution failure");
        }
    }
}

Gpu3dFft::Gpu3dFft(const PmeGpu*        pmeGpu,
                   ivec                 realGridSize,
                   ivec                 realGridSizePadded,
                   ivec                 complexGridSize,
                   ivec                 complexGridSizePadded,
                   const bool           useDecomposition,
                   const bool           performOutOfPlaceFFT,
                   const DeviceContext& context,
                   const DeviceStream&  pmeStream,
                   DeviceBuffer<float>  realGrid,
                   DeviceBuffer<float>  complexGrid,
                   DeviceBuffer<float>  complexGrid2) :
    impl_(std::make_unique<Impl>(pmeGpu,
                                 realGridSize,
                                 realGridSizePadded,
                                 complexGridSize,
                                 complexGridSizePadded,
                                 useDecomposition,
                                 performOutOfPlaceFFT,
                                 context,
                                 pmeStream,
                                 realGrid,
                                 complexGrid,
                                 complexGrid2))
{
}

Gpu3dFft::~Gpu3dFft() = default;

} // namespace gmx
