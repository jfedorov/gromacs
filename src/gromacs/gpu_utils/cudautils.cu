/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2014,2015,2016,2017 The GROMACS development team.
 * Copyright (c) 2018,2019,2020, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

#include "gmxpre.h"

#include "cudautils.cuh"

#include <cassert>
#include <cstdlib>

#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/utility/gmxassert.h"

/*** Generic CUDA data operation wrappers ***/

// TODO: template on transferKind to avoid runtime conditionals
int cu_copy_D2H(void* h_dest, void* d_src, size_t bytes, GpuApiCallBehavior transferKind, hipStream_t s = nullptr)
{
    hipError_t stat;

    if (h_dest == nullptr || d_src == nullptr || bytes == 0)
    {
        return -1;
    }

    switch (transferKind)
    {
        case GpuApiCallBehavior::Async:
            GMX_ASSERT(isHostMemoryPinned(h_dest), "Destination buffer was not pinned for CUDA");
            stat = hipMemcpyAsync(h_dest, d_src, bytes, hipMemcpyDeviceToHost, s);
            CU_RET_ERR(stat, "DtoH hipMemcpyAsync failed");
            break;

        case GpuApiCallBehavior::Sync:
            stat = hipMemcpy(h_dest, d_src, bytes, hipMemcpyDeviceToHost);
            CU_RET_ERR(stat, "DtoH hipMemcpy failed");
            break;

        default: throw;
    }

    return 0;
}

int cu_copy_D2H_sync(void* h_dest, void* d_src, size_t bytes)
{
    return cu_copy_D2H(h_dest, d_src, bytes, GpuApiCallBehavior::Sync);
}

/*!
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
int cu_copy_D2H_async(void* h_dest, void* d_src, size_t bytes, hipStream_t s = nullptr)
{
    return cu_copy_D2H(h_dest, d_src, bytes, GpuApiCallBehavior::Async, s);
}

// TODO: template on transferKind to avoid runtime conditionals
int cu_copy_H2D(void* d_dest, const void* h_src, size_t bytes, GpuApiCallBehavior transferKind, hipStream_t s = nullptr)
{
    hipError_t stat;

    if (d_dest == nullptr || h_src == nullptr || bytes == 0)
    {
        return -1;
    }

    switch (transferKind)
    {
        case GpuApiCallBehavior::Async:
            GMX_ASSERT(isHostMemoryPinned(h_src), "Source buffer was not pinned for CUDA");
            stat = hipMemcpyAsync(d_dest, h_src, bytes, hipMemcpyHostToDevice, s);
            CU_RET_ERR(stat, "HtoD hipMemcpyAsync failed");
            break;

        case GpuApiCallBehavior::Sync:
            stat = hipMemcpy(d_dest, h_src, bytes, hipMemcpyHostToDevice);
            CU_RET_ERR(stat, "HtoD hipMemcpy failed");
            break;

        default: throw;
    }

    return 0;
}

int cu_copy_H2D_sync(void* d_dest, const void* h_src, size_t bytes)
{
    return cu_copy_H2D(d_dest, h_src, bytes, GpuApiCallBehavior::Sync);
}

/*!
 *  The copy is launched in stream s or if not specified, in stream 0.
 */
int cu_copy_H2D_async(void* d_dest, const void* h_src, size_t bytes, hipStream_t s = nullptr)
{
    return cu_copy_H2D(d_dest, h_src, bytes, GpuApiCallBehavior::Async, s);
}
